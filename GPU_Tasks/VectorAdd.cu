#include "hip/hip_runtime.h"
#include<stdio.h>

#define N 200





__global__ void addvector(int* a,int* b,int* c)
    {
        int x=threadIdx.x;
        
        if(x<N)
        {
        c[x]=a[x]+b[x];
        }


    }








int main()
{
    int ch,arr[N],brr[N],result[N],*gpu1,*gpu2,*res;
    
    printf("Enter A Number Between 1 and 200 : ");
    scanf("%d",&ch);
    
    ch=(ch>N)?N:ch;
    
    printf("Enter Numbers Of First Array : \n");
    
    int i;
        for(i=0;i<ch;i++)
        {
            scanf("%d",&arr[i]);
           
        }
        
   printf("Enter Numbers Of Second Array : \n");     
        
        for(i=0;i<ch;i++)
        {
            scanf("%d",&brr[i]);
        }            
    
        for(i=ch;i<N;i++)
        {
        arr[i]=0;
        brr[i]=0;
        }
    
    
   // dim3 numBlocks(1,1);
   // dim3 numThreads(N,1);
    
    
        
    hipMalloc((void**)&gpu1,N * sizeof(int));    
    hipMalloc((void**)&gpu2,N * sizeof(int));
    hipMalloc((void**)&res,N * sizeof(int));
    
    hipMemcpy(gpu1,arr,N * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gpu2,brr,N * sizeof(int),hipMemcpyHostToDevice);
    
    
    
    addvector<<<1,N>>>(gpu1,gpu2,res);
    hipDeviceSynchronize();
   
    hipMemcpy(result,res,N * sizeof(int),hipMemcpyDeviceToHost);
    
    
    
    
       printf("The Result Of Addiing The Array Elements Are : \n");
       
       for(i=0;i<ch;i++)
        {
            printf("%d\t",result[i]);
           
        }
    printf("\n");    
        
    hipFree(gpu1);
    hipFree(gpu2);
    hipFree(res);
        
    
   return 0; 

}
