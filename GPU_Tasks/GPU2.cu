#include "hip/hip_runtime.h"
#include<iostream>


/*May Change it Later.
 Assuming 6 Variables
 Copy it only once in memory for processing
 Using simple Malloc Function
*/ 
#define height 640
#define width 480
#define isize 307200
#define N 192
#define bsize 100

/*Store Pixel Density in CUDA Memory for Analysis */
int pdensity[N],*gthresh; 


//I will get The Thread ID From The Called Function
/* 

    Swan Dive Into Zero-Copy MEMORY World 
    Keeping My Fingers Crossed
    gpuptr stores the gpu location of mapped memory
    cpuptr stores the cpu location of mapped memory
*/
unsigned char *gpuptr,*cpuptr,*fcount,*gpucount;

__const__ curve_data[200]; //To allocate fast memory

typedef struct block
{
    int height;
    int width;
    unsigned char data[bsize][bsize]; //40x40 image
}Blocks;

typedef struct Point
{
    int x;
    int y;

}Pt;

/*Assign Blocks in CUDA Memory for fast execution*/

Blocks *small_image;
Pt *loc;

void assign_threshold(int t[])
{
        hipMalloc(pdensity,N*sizeof(int));
        hipMemset(pdensity,0,N*sizeof(int));
        hipMalloc(loc,sizeof(Pt));
        hipMalloc(gpucount,sizeof(int));
        hipMalloc(small_image,N * sizeof(Blocks));
        hipMemcpy(gthresh,t,6 * sizeof(int),hipMemcpyHostToDevice);
}      


__global__ void cleanup()
{
    hipHostFree(cpuptr);
    hipFree(small_image);
    hipFree(gthresh);
}


char* MapImageToCPU()
{
    //Add Finger Count Here As Well
    char* ptr=NULL;
          fcount=NULL;
    
    hipSetDeviceFlags(hipDeviceMapHost); 
    
    /*Assign Both CPU image and GPU Count*/
    
    hipHostAlloc(&ptr,height*width,hipHostMallocMapped);
    hipHostAlloc(&fcount,1,hipHostMallocMapped);
    
    /*Assign The GPU Pointer To The Location Made by HostAlloc
                        In CPU Memory */
    
    hipHostGetDevicePointer(&gpuptr,ptr);
    hipHostGetDevicePointer(&gpucount,fcount);
    
    cpuptr=ptr;
    return ptr;
} 

void* MapVariable(int size)
{
    
    //Add Finger Count Here As Well
    void* ptr=NULL;
          fcount=NULL;
    
    hipSetDeviceFlags(hipDeviceMapHost); 
    /*Assign Both CPU image and GPU Count*/
    hipHostAlloc(&ptr,height*width,hipHostMallocMapped);
    return ptr;
}

void* GetGPUAddress(int* &cpuid)
{
    void* ptr=NULL;
    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostGetDevicePointer(&ptr,cpuid);
    return ptr;
}

 
 /* I'm Calling Split Blocks According To Symmetric Blocks And Threads  */
 
__global__ void splitblocks() 
{                                                           
 /* Format of dim3 is (x,y,z)
    If we assign (1,200) that means 1 grid and 200 blocks 
    So
    x=1
    y=200
 */   
 
  int bID=blockIdx.y;
  int x=threadIdx.x;
  int y=threadIdx.y;
  
  int pixel_loc = int(&gpuptr)+((bID*blockDim.y)+y); // We Don't need x here because block ID gives X-coordinate value
  
  /* I'm not doing this in a single loop because all threads might access the same 
     pixel density variable, which might lead to Use of Extra atomic Functions
     So, I separated The Image into a grid for easy access
  */   
  
    if((bID<=N) && (
     {
            small_image[bID]->data[x][y]=gpuptr[pixel_loc];
     } 
}


__global__ void write_density()
    {
        int x=threadIdx.x;
        pdensity[x]=0;
        int i,j;
        for(i=0;i<N;i++)
        {
            for(j=0;j<N;j++)
            {
               if(small_image[x]->data[i][j]==255)
               {
                 pdensity[x]++; //This addition is threadsafe
               }
            
            }
        
        }

    }

int fingers(unsigned char* image)
{
/*Assuming I've Already Have Assigned CPU Image Mapped Pointer */

        int max=0;
        int b=0;
        int res=0;
        /*Searching Function*/
        dim3 sblocks (1,N);
        dim3 sthreads (bsize,bsize); // NxN array
        
        splitblocks <<< sblocks,sthreads >>> ();
              
        
        
        /* Adding Function */
 
        write_density <<< 1,200 >>> ();
 
            
        /*Synchronize All working threads*/
            
        hipDeviceSynchronize();
        
        /* Simple Max Function after Doing Everything */     
        for(i=0;i<N;i++)
        {
           if(max<pdensity[i])
           {
                max=pdensity[i];
                b=i;
           }    
        
         }      
        

return res;

}

/*Block Thread Process */

__global__ checkarc(bool direction)
{
       int bID= BlockIdx.y;
       int tx=threadIdx.x;
       

       if(direction == 1) // From Left To Right
       {
          if (small_image[bID].data[circle_y(tx)][tx] == 1)
          {
                    loc.x=//I don't know
          
          }       
       
       
       }
        
       else
       {
             
        
        
        
       }


}


__device__ int circle_y(int x)
{
   return (int)(sqrt(float((N*N)-(x*x))));

}
