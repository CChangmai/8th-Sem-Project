#include "hip/hip_runtime.h"
#include<iostream>


/*May Change it Later.
 Assuming 6 Variables
 Copy it only once in memory for processing
 Using simple Malloc Function
*/ 
#define height 640
#define width 480
#define isize 307200
#define N 192
#define bsize 100

/*Store Pixel Density in CUDA Memory for Analysis */
int pdensity[N],*gthresh; 


//I will get The Thread ID From The Called Function
/* 

    Swan Dive Into Zero-Copy MEMORY World 
    Keeping My Fingers Crossed
    gpuptr stores the gpu location of mapped memory
    cpuptr stores the cpu location of mapped memory
*/
unsigned char *gpuptr,*cpuptr,*fcount,*gpucount;


/*unsigned char* curve_data;*/ //To See Pixel Location

typedef struct block
{
    int height;
    int width;
    unsigned char data[bsize][bsize]; //40x40 image
}Blocks;


/*
    Assign Blocks in CUDA Memory for fast execution
    Blocks Is User-Defined Datatype defined above
*/

Blocks *small_image;
/*
Use This If U need To Find The Cursor Location
int *loc_x,*loc_y; 
*/
 


__global__ void cleanup()
{
    //hipHostFree(cpuptr); Not Doing This due to some problems in main function
    hipFree(small_image);
    
}


unsigned char* MapImageToCPU()
{
    //Add Finger Count Here As Well
    unsigned char* ptr=NULL;
          fcount=NULL;
    
    hipSetDeviceFlags(hipDeviceMapHost); 
    
    /*Assign Both CPU image and GPU Count*/
    
    hipHostAlloc(&ptr,height*width,hipHostMallocMapped);
    hipHostAlloc(&fcount,1,hipHostMallocMapped);
    
    /*Assign The GPU Pointer To The Location Made by HostAlloc
                        In CPU Memory */
    
    hipHostGetDevicePointer(&gpuptr,ptr);
    hipHostGetDevicePointer(&gpucount,fcount);
    
    cpuptr=ptr;
    return ptr;
} 

void* MapVariable(int size)
{
    
    //Add Finger Count Here As Well
    void* ptr=NULL;
      
    
    hipSetDeviceFlags(hipDeviceMapHost); 
    /*Assign Both CPU image and GPU Count*/
    hipHostAlloc(&ptr,size,hipHostMallocMapped);
    
    return ptr;
}

unsigned char* MapFilter()
{
    
    //Add Finger Count Here As Well
    unsigned char* ptr=NULL;
    hipSetDeviceFlags(hipDeviceMapHost); 
    /*Assign Both CPU image and GPU Count*/
    hipHostAlloc(&ptr,bsize*bsize,hipHostMallocMapped);
    hipHostGetDevicePointer(&filter,ptr);
    return ptr;
}

void* GetGPUAddress(int* &cpuid)
{
    void* ptr=NULL;
    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostGetDevicePointer(&ptr,cpuid);
    return ptr;
}

/*
GET A POINTER TO CPU_MEMORY TO ASSIGN THRESHOLD RATHER THAN COPYING ANYTHING TO GPU
*/
int* assign_threshold()
{
        
        hipMalloc(pdensity,N*sizeof(int));
        hipMemset(pdensity,0,N*sizeof(int));
        //hipMalloc(loc,sizeof(Pt));
        hipMalloc(gpucount,sizeof(int));
        hipMalloc(small_image,N * sizeof(Blocks));
     
     
     //-----------NOW MAP DEVICE MEMORY---------------------------
     
     int* ptr=NULL;
     hipSetDeviceFlags(hipDeviceMapHost); 
     /*Assign Both CPU image and GPU Count*/
     hipHostAlloc(&ptr,5*sizeof(int),hipHostMallocMapped);
     hipHostGetDevicePointer(&gthresh,ptr);
     return *ptr;
}     
 
 /* I'm Calling Split Blocks According To Symmetric Blocks And Threads  */
 
__global__ void splitblocks() 
{                                                           
 /* Format of dim3 is (x,y,z)
    If we assign (1,200) that means 1 grid and 200 blocks 
    So
    x=1
    y=200
 */   
 
  int bID=blockIdx.y;
  int x=threadIdx.x;
  int y=threadIdx.y;
  
  int pixel_loc = int(&gpuptr)+((bID*blockDim.y)+y); // We Don't need x here because block ID gives X-coordinate value
  
  /* I'm not doing this in a single loop because all threads might access the same 
     pixel density variable, which might lead to Use of Extra atomic Functions
     So, I separated The Image into a grid for easy access
  */   
  
    if((bID<=N) && ((x*y)<=isize)
     {
            small_image[bID]->data[x][y]=gpuptr[pixel_loc];
     } 
}


__global__ void write_density()
    {
        int x=threadIdx.x;
        pdensity[x]=0;
        int i,j;
        for(i=0;i<N;i++)
        {
            for(j=0;j<N;j++)
            {
               if(small_image[x]->data[i][j]==1)
               {
                 pdensity[x]++; //This addition is threadsafe
               }
            
            }
        
        }

    }

int finger_location()
{
/*Assuming I've Already Have Assigned CPU Image Mapped Pointer */

        int BLOCK=0;
        /*Searching Function*/
        dim3 sblocks (1,N);
        dim3 sthreads (bsize,bsize); // NxN array
        
        splitblocks <<< sblocks,sthreads >>> ();
              
        
        
        /* Adding Function */
 
        write_density <<< 1,200 >>> ();
 
            
        /*Synchronize All working threads*/
            
        hipDeviceSynchronize();
        
        /* Simple CALLING Function after Doing Everything */     
        for(i=0;i<N;i++)
        {
           if(pdensity[i]<gthresh[0])
           {
              BLOCK=i;
           }    
        
         }      
        
cleanup();
return BLOCK;

}

void CleanImage()
{
hipHostFree(cpuptr);
}
/*Block Thread Process 

Unfinished Cursor Finding Process

__global__ checkarc(bool direction)
{
       int bID= BlockIdx.y;
       int tx=threadIdx.x;
       

       if(direction == 1) // From Left To Right
       {
          if (small_image[bID].data[circle_y(tx)][tx] == 1)
          {
                    loc.x=                          //I don't know
          
          }       
       
       
       }
        
       else
       {
             
        
        
        
       }



}
*/


/*Dummy Main Function
  Just To Be On The Safe Side
 */

